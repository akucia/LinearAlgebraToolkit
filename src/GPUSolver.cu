#include "hip/hip_runtime.h"
/*
 * GPUSolver.cpp
 *
 *      Author: Artur Kucia
 */

#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstring>
#include <cfloat>
#include "GPUSolver.cuh"
#include "Matrix.h"
#include "Vector.h"



GPUSolver::GPUSolver() {

}

GPUSolver::~GPUSolver() {

}


__global__ void kernel_jj(float *d_a, float *d_i, int n, int j){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if( i < n){
		if ( d_a[j*n+j] == 0.0)
			for(int k=j+1; k<n; k++)
				if ( d_a[k*n+j] != 0.0){
					d_a[j*n+i] += d_a[k*n+i];
					d_i[j*n+i] += d_i[k*n+i];
					break;
				}

	}

}


__global__ void normalize_row(float *d_a, float *d_i, int n, int j){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n){
		if (i !=j){
			d_i[j*n+i] /= d_a[j*n+j];
			d_a[j*n+i] /= d_a[j*n+j];
		}
		else{
			d_i[j*n+i] /= d_a[j*n+j];
		}
	}

}

__global__ void normalize_diagonal(float *d_a, float *d_i, int n, int j){

		d_a[j*n+j] /= d_a[j*n+j];


}

__global__ void reduce_row_i(float *d_a, float *d_i, int n, int j){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(x<n && y< n)
	if (y != j){
		d_i[y*n+x] -= d_i[j*n+x]*d_a[y*n+j];
	}

}
__global__ void reduce_row_a(float *d_a, float *d_i, int n, int j){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(x<n && y< n)
	if (y != j){
		if( x != j)
		d_a[y*n+x] -= d_a[j*n+x]*d_a[y*n+j];


	}

}


Matrix GPUSolver::solveGJ(Matrix A, Vector B) {

	Matrix A_copy(A);

	int n = A.getN();
	int m = B.getM();

	Matrix I(n);
	float *h_i = I.getArrayPointer();
	float *h_a = A.getArrayPointer();


	float *d_a;
	float *d_i;

	hipMalloc((void **)&d_a, n*n*sizeof(float));
	hipMalloc((void **)&d_i, n*n*sizeof(float));

	hipMemcpy(d_a, h_a, n*n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_i, h_i, n*n*sizeof(float), hipMemcpyHostToDevice);


	for(int i=0; i<n; i++){

		dim3 block(256,1);
		dim3 grid((n + block.x - 1) / block.x, 1);
		kernel_jj <<<grid, block>>> (d_a, d_i, n, i);


		dim3 block1(256,1);
		dim3 grid1((n + block1.x - 1) / block1.x,1);
		normalize_row <<< grid1,block1 >>> (d_a, d_i, n, i);

		dim3 block2(16,16);
		dim3 grid2((n + block2.x - 1) / block2.x, (n + block2.y - 1) / block2.y);
		reduce_row_i <<< grid2, block2 >>> (d_a, d_i, n, i);

		reduce_row_a <<< grid2, block2 >>> (d_a, d_i, n, i);

	}


	hipMemcpy(h_a, d_a, n*n*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_i, d_i, n*n*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_i);

	return I.matMul(B);
}

__global__ void kernel_update_u(float *d_a, int n, int k){

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	float sum = 0.0;
	if(i>k-1 && i<n){
		for(int p=0;p<k;++p){

			sum += d_a[i*n+p]*d_a[p*n+k];
		}

	d_a[i*n+k] -= sum;

	}




}
__global__ void kernel_update_l(float *d_a, int n, int k){

	int j = blockIdx.x * blockDim.x + threadIdx.x;

	float sum=0.0;
	if(j>k && j<n){
		for(int p=0;p<k;++p){

			sum+=d_a[k*n+p]*d_a[p*n+j];
		}
		d_a[k*n+j]=(d_a[k*n+j]-sum)/d_a[k*n+k];
	}

}

__global__ void kerneljj(float *d_a, float *d_b, int n, int j){

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < n){
		if ( d_a[j*n+j] == 0.0){
			for(int k=j+1; k<n; k++){

				if (d_a[k*n+j] != 0.0){
					d_a[j*n+i] += d_a[k*n+i];
					if(i==0){
						d_b[j] += d_b[k];
					}
					break;
				}
			}
		}
	}

}



Matrix GPUSolver::solveLUD(Matrix A, Vector B){

	int n = A.getN();
	int m = B.getM();
	Vector X(n);

	float *h_a = A.getArrayPointer();
	float *h_b = B.getArrayPointer();
	float *h_x = X.getArrayPointer();
	float *h_y = new float[n];

	for(int i=0; i<n; i++)
		h_y[i] = 0.0;

	float *d_a;
	float *d_b;


	hipMalloc((void **)&d_a, n*n*sizeof(float));
	hipMalloc((void **)&d_b, n*sizeof(float));


	hipMemcpy(d_a, h_a, n*n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, n*sizeof(float), hipMemcpyHostToDevice);


	for(int k=0;k<n;++k){

		dim3 block2(256,1);
		dim3 grid2((n + block2.x - 1) / block2.x,1);

		kerneljj <<< grid2, block2 >>> (d_a, d_b, n, k);
		kernel_update_u <<< grid2, block2 >>> (d_a, n, k);
		kernel_update_l <<< grid2, block2 >>> (d_a, n, k);

	}


	hipMemcpy(h_a, d_a, n*n*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_b, d_b, n*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);

	for(int i=0;i<n;++i){

		float sum = 0.0;
		for(int k=0;k<i;++k){

			sum += h_a[i*n+k]*h_y[k];
		}
		h_y[i]=(h_b[i]-sum)/h_a[i*n+i];
	}
	for(int i=n-1; i>=0 ;--i){

		float sum = 0.0;
		for(int k=i+1; k<n; ++k){

			sum += h_a[i*n+k]*h_x[k];
		}
		h_x[i] = (h_y[i]-sum);
	}

	delete [] h_y;
	return X;
}
